#include "hip/hip_runtime.h"
#include "../include/cuda_example.cuh"
#include <stdio.h>
#include <string.h>

/**
 * @brief Says hello from the GPU.
 */
__global__ void HelloGpuKernel(char* gpuName)
{
    printf("%s says: Hello, World!\n", gpuName);
}

/**
 * @brief Tells the GPU to say hello.
 */
void HelloGpu()
{
    hipError_t error;

    /* Try to locate an eligible GPU. */
    int deviceNumber;
    error = hipGetDevice(&deviceNumber);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failure finding a NVIDIA GPU: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    /* Try to get the said GPU's device properties. */
    hipDeviceProp_t gpu;
    error = cudaGetDeviceProperties_v2(&gpu, deviceNumber);
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failure getting gpu%d's device properties: %s\n", deviceNumber, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    /* Copy the gpu's name to the device. */
    char* gpuName;
    hipMalloc(&gpuName, sizeof(gpu.name));
    hipMemcpy(gpuName, gpu.name, sizeof(gpu.name), hipMemcpyHostToDevice);

    /* Say hello. */
    HelloGpuKernel << <1, 1 >> > (gpuName);
    hipDeviceSynchronize();

    /* Make sure everything is good! */
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        error == cudaErrorUnsupportedPtxVersion
            ? fprintf(stderr, "Change the `CMAKE_CUDA_ARCHITECTURES` in the CMakeLists file!\n")
            : fprintf(stderr, "Failure calling HelloGpuKernel: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    /* Free the gpu's name. */
    hipFree(gpuName);
}